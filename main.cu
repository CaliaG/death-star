#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h> // Include so GLM picks up the compiler version
#define GLM_FORCE_CUDA

#include "camera.h"
#include "hittable.h"
#include "template_scenes.h"
#include "bvh.h"
#include "loadOBJ.h"
#include "render.cu"

#include <iostream>
#include <fstream>
#include <cstdlib>

#include <hiprand/hiprand_kernel.h>

#define STB_IMAGE_IMPLEMENTATION
#include "libs/stb/stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "libs/stb/stb_image_write.h"

// Since memory for hittables must already be allocated when creating
// them on the GPU, I currently store a static number of how many hittables
// are manually created - num_manually_defined_hittables.
// It is far less than ideal, and a potential workaround would be to instead
// of creating them directly on the GPU, I create a bunch of sphereData and
// triangleData structs, similar to the ones I create for .obj files, which
// are stored on the CPU, so we can use their count to allocate the correct
// amount of memory for hittables, before sending them off the the GPU to be created.
//
// The reason I am not fully keen on that, is that we'll have an extra step
// and also a copy for each sphere and triangle on the CPU, which seems wasteful
//
// Additionally, most high profile renderers, have their own file formats (
// Arnold .ass, Renderman RIB, etc.) that describe a scene, which contain
// the number of objects to render, so in those cases, the number of hittables
// is always known, so there is no need for neither the above mentioned proceedure
// nor the following manually maintained static value.
__global__
void manually_populate_scene(Hittable* hittables, int start_id, hiprandState* rand_state)
{
#define num_manually_defined_hittables 3
	hittables[start_id+0] = Hittable::sphere(vec3(0,-1000,0), 1000,
			Material::lambertian(vec3(0.2, 0.2, 0.35)));
	hittables[start_id+1] = Hittable::sphere(vec3(0,.5,0), .5,
			Material::metal(vec3(.5, .5, .5), .0));
	hittables[start_id+2] = Hittable::sphere(vec3(.7,.25,0), .25,
			Material::dielectric(1.5));
}

void createScene(Scene& scene, hiprandState* rand_state) {
    objData obj = load_obj("/content/death-star/models/bunny.obj");
    objData obj2 = load_obj("/content/death-star/models/monkey.obj");
	// scene.num_hittables = obj.num_triangles + num_manually_defined_hittables;
	scene.num_hittables = obj.num_triangles + obj2.num_triangles + num_manually_defined_hittables;

	hipMalloc(&(scene.hittables), scene.num_hittables * sizeof(Hittable));

	Material* material;
	hipMalloc(&(material), sizeof(Material));
	//create_metal<<<1, 1>>>(material, vec3(.1, .3, .5), .5);
	//create_metal<<<1, 1>>>(material, rand_state);
	//create_lambertian<<<1, 1>>>(material, vec3(.5, .1, .45));
	create_dielectric<<<1, 1>>>(material, 1.5f);

	Material* material2;
	hipMalloc(&(material2), sizeof(Material));
	create_metal<<<1, 1>>>(material2, vec3(.1, .3, .5), .5);

	int obj_threads = 512;
	int obj_dims = (obj.num_triangles + obj_threads - 1) / obj_threads;
	create_obj_hittables<<<obj_dims, obj_threads>>>(scene.hittables, material, obj, 0, 0.8f);

	obj_dims = (obj2.num_triangles + obj_threads - 1) / obj_threads;
	create_obj_hittables<<<obj_dims, obj_threads>>>(scene.hittables, material2, obj2, obj.num_triangles, 0.5f);

	// manually_populate_scene<<<1, 1>>>(scene.hittables, obj.num_triangles, rand_state);
	manually_populate_scene<<<1, 1>>>(scene.hittables, obj.num_triangles + obj2.num_triangles, rand_state);
}

void save_to_ppm(float *fb, int nx, int ny) {
    std::ofstream ofs;
    ofs.open("./image.ppm", std::ios::out | std::ios::binary);
    ofs << "P3\n" << nx << " " << ny << "\n255\n";
        for (int j = ny-1; j >= 0; j--) {
            for (int i = 0; i < nx; i++) {
                size_t pixel_index = j*nx + i;
                int ir = int(255.99*fb[pixel_index * 3 + 0]);
                int ig = int(255.99*fb[pixel_index * 3 + 1]);
                int ib = int(255.99*fb[pixel_index * 3 + 2]);
                ofs << ir << " " << ig << " " << ib << "\n";
            }
        }
    ofs.close();
}

void save_to_jpg(float *fb, int nx, int ny) {
    uint8_t* imgBuff = (uint8_t*)std::malloc(nx * ny * 3 * sizeof(uint8_t));
    for (int j = ny - 1; j >= 0; --j) {
        for (int i = 0; i < nx; ++i) {
            size_t index = j * nx + i;
            // -- stbi generates a Y flipped image
            size_t rev_index = (ny - j - 1) * nx + i;
            float r = fb[index * 3 + 0];
            float g = fb[index * 3 + 1];
			float b = fb[index * 3 + 2];
            imgBuff[rev_index * 3 + 0] = int(255.999f * r) & 255;
            imgBuff[rev_index * 3 + 1] = int(255.999f * g) & 255;
            imgBuff[rev_index * 3 + 2] = int(255.999f * b) & 255;
        }
    }
    //stbi_write_png("out.png", nx, ny, 3, imgBuff, nx * 3);
    stbi_write_jpg("image.jpg", nx, ny, 3, imgBuff, 100);
    std::free(imgBuff);
}

int main(int argc, char** argv) {
	int width = 1920;
	int height = 1080;
	int num_samples = 100;
	int max_bounces = 8;

	printf("Initializing death-star for %ix%i pixels, %i samples and %i max bounces\n",
			width, height, num_samples, max_bounces);

	// Calculate blocks and threads
	int tx = 8, ty = 8; // bucket size
	
    clock_t start, stop;
	start = clock();
	
	dim3 blocks(width/tx + 1, height/ty + 1);
	dim3 threads(tx, ty);

	// CUDA random number generator
	hiprandState *rand_state;
	hipMalloc((void**)&rand_state, (width * height) * sizeof(hiprandState));

	// Camera
	Camera* camera;
	hipMalloc(&camera, 1 * sizeof(Camera));

	initialize_renderer<<<blocks, threads>>>(width, height, rand_state);
	initialize_camera<<<1, 1>>>(camera, vec3(-.253,1.731,7.573), vec3(-.253,1.119,.281),
			vec3(0,1,0), 20, float(width)/float(height), 0.1, 7.317);

	// Create scene
	Scene scene;
	createScene(scene, rand_state);

	// Create BVH
	BVHNode* bvh_root = create_BVH(scene.hittables, scene.num_hittables);

	// Allocate memory for pixels
	float *pixel_buffer, *d_pixel_buffer;
	pixel_buffer = (float*)malloc(width * height * 3 * sizeof(float));
	hipMalloc(&d_pixel_buffer, width * height * 3 * sizeof(float));

	// Render into buffer
	render<<<blocks, threads>>>(width, height, num_samples, max_bounces, d_pixel_buffer,
			bvh_root, rand_state, camera);

	// Copy pixel data from device to cpu
	hipMemcpy(pixel_buffer, d_pixel_buffer,
			width * height * 3 * sizeof(float), hipMemcpyDeviceToHost);

    stop = clock();
    double timer_seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
    std::cout << "took " << timer_seconds << " seconds.\n";

	//save_to_ppm(pixel_buffer, width, height);
	save_to_jpg(pixel_buffer, width, height);

	return 0;
}
